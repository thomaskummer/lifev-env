/*
//@HEADER
// ************************************************************************
// 
//   KokkosArray: Manycore Performance-Portable Multidimensional Arrays
//              Copyright (2012) Sandia Corporation
// 
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions? Contact  H. Carter Edwards (hcedwar@sandia.gov) 
// 
// ************************************************************************
//@HEADER
*/

#include <TestStochastic.hpp>

#include <KokkosArray_Cuda.hpp>
#include <Host/KokkosArray_Host_ProductTensor.hpp>
#include <Cuda/KokkosArray_Cuda_SymmetricDiagonalSpec.hpp>
#include <Cuda/KokkosArray_Cuda_ProductTensor.hpp>
#include <Cuda/KokkosArray_Cuda_CrsProductTensorLegendre.hpp>
#include <Cuda/KokkosArray_Cuda_StochasticProductTensor.hpp>
#include <Cuda/KokkosArray_Cuda_BlockCrsMatrix.hpp>
#include <Cuda/KokkosArray_Cuda_CrsMatrix.hpp>

namespace unit_test {

template<typename Scalar>
struct performance_test_driver<Scalar,KokkosArray::Cuda> {
  static void run(bool test_flat, bool test_orig, bool test_block) {
    typedef KokkosArray::Cuda Device;
    
    int nGrid;
    int nIter; 

    // All methods compared against flat-original
    if (test_flat) {
      nGrid = 5 ;
      nIter = 1 ; 
      performance_test_driver_all<Scalar,Device>( 
	3 , 1 ,  9 , nGrid , nIter , test_block );
      performance_test_driver_all<Scalar,Device>( 
	5 , 1 ,  5 , nGrid , nIter , test_block );
    }
    
    // Just polynomial methods compared against original
    if (test_orig) {
      nGrid = 32 ;
      nIter = 1 ; 
      performance_test_driver_poly<Scalar,Device>( 
	3 , 1 , 12 , nGrid , nIter , test_block );
      performance_test_driver_poly<Scalar,Device>( 
	5 , 1 ,  6 , nGrid , nIter , test_block );
    }
    
  }

};

}

template <typename Scalar>
int mainCuda(bool test_flat, bool test_orig, bool test_block, int device_id)
{
  typedef unsigned long long int IntType ;

  KokkosArray::Cuda::initialize( KokkosArray::Cuda::SelectDevice(0) );

  hipSetDevice(device_id);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device_id);
  std::cout << std::endl 
	    << "Device " << device_id << ": " << deviceProp.name 
	    << std::endl;

  std::cout << std::endl << "\"Cuda Performance\"" << std::endl ;
  unit_test::performance_test_driver<Scalar,KokkosArray::Cuda>::run(
    test_flat, test_orig, test_block);

  KokkosArray::Cuda::finalize();

  return 0 ;
}

template int mainCuda<float>(bool, bool, bool, int);
template int mainCuda<double>(bool, bool, bool, int);
