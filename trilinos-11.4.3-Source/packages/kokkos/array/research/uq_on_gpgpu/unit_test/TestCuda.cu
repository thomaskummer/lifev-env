/*
//@HEADER
// ************************************************************************
// 
//   KokkosArray: Manycore Performance-Portable Multidimensional Arrays
//              Copyright (2012) Sandia Corporation
// 
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions? Contact  H. Carter Edwards (hcedwar@sandia.gov) 
// 
// ************************************************************************
//@HEADER
*/

#include <KokkosArray_Host.hpp>
#include <KokkosArray_Cuda.hpp>

#include <KokkosArray_LegendrePolynomial.hpp>
#include <KokkosArray_SymmetricDiagonalSpec.hpp>
#include <KokkosArray_BlockCrsMatrix.hpp>
#include <KokkosArray_CrsMatrix.hpp>

//

#include <Cuda/KokkosArray_Cuda_SymmetricDiagonalSpec.hpp>
#include <Cuda/KokkosArray_Cuda_CrsProductTensorLegendre.hpp>
#include <Cuda/KokkosArray_Cuda_SparseProductTensorLegendre.hpp>
#include <Cuda/KokkosArray_Cuda_BlockCrsMatrix.hpp>
#include <Cuda/KokkosArray_Cuda_CrsMatrix.hpp>

//

#include <TestBlockCrsMatrix.hpp>
#include <TestStochastic.hpp>

namespace unit_test {

template<typename Scalar>
struct performance_test_driver<Scalar,KokkosArray::Cuda> {
  static void run(bool test_flat, bool test_orig, bool test_block, bool check){
    typedef KokkosArray::Cuda Device;
    
    int nGrid;
    int nIter; 
    bool print;

    // All methods compared against flat-original
    if (test_flat) {
      nGrid = 5 ;
      nIter = 1 ; 
      print = false ;
      performance_test_driver_all<Scalar,Device>( 
	3 , 1 ,  9 , nGrid , nIter , print , test_block , check );
      performance_test_driver_all<Scalar,Device>( 
	5 , 1 ,  5 , nGrid , nIter , print , test_block , check );
    }
    
    // Just polynomial methods compared against original
    if (test_orig) {
      nGrid = 32 ;
      nIter = 1 ; 
      print = false ;
      performance_test_driver_poly<Scalar,Device>( 
	3 , 1 , 12 , nGrid , nIter , print , test_block , check );
      performance_test_driver_poly<Scalar,Device>( 
	5 , 1 ,  6 , nGrid , nIter , print , test_block , check );
    }
    
    //------------------------------
    
    /*
      std::cout << std::endl
      << "\"CRS flat-matrix ~27 nonzeros/row (CUDA uses cusparse)\""
      << std::endl
      << "\"nGrid\" , "
      << "\"VectorSize\" , "
      << "\"MXV-Time\""
      << std::endl ;
      
      for ( int n_grid = 10 ; n_grid <= 100 ; n_grid += 5 ) {
      
      const std::pair<size_t,double> perf_flat =
      test_flat_matrix<double,Device>( n_grid , nIter , print );
      
      std::cout << n_grid << " , "
      << perf_flat.first << " , "
      << perf_flat.second
      << std::endl ;
      }
    */
    
    //------------------------------
  }

};

}

template <typename Scalar>
int mainCuda(bool test_flat, bool test_orig, bool test_block, bool check, 
	     int device_id)
{
  typedef unsigned long long int IntType ;

  KokkosArray::Cuda::initialize( KokkosArray::Cuda::SelectDevice(0) );

  hipSetDevice(device_id);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device_id);
  std::cout << std::endl 
	    << "Device " << device_id << ": " << deviceProp.name 
	    << std::endl;

  typedef KokkosArray::Cuda device ;

  //------------------------------------
  // Quick correctness check:

  const std::vector<int> var( 3 , 2 ); // #Stochastic variables = 3 , polynomical degree = 2
  const int ngrid = 3 ; // 3x3x3 element grid

  unit_test::test_product_flat_original_matrix<  float, device>( var , ngrid , 1 , true );
  unit_test::test_product_flat_original_matrix<  double,device>( var , ngrid , 1 , true );

  unit_test::test_product_flat_commuted_matrix<  float, device>( var , ngrid , 1 , true );
  unit_test::test_product_flat_commuted_matrix<  double,device>( var , ngrid , 1 , true );

  unit_test::test_product_tensor_diagonal_matrix<float, device>( var , ngrid , 1 , true );
  unit_test::test_product_tensor_diagonal_matrix<double,device>( var , ngrid , 1 , true );

  unit_test::test_product_tensor_legendre< KokkosArray::CrsProductTensorLegendre< float , device > , float , float >( var , ngrid , 1 , true );
  unit_test::test_product_tensor_legendre< KokkosArray::CrsProductTensorLegendre< double, device > , double, double>( var , ngrid , 1 , true );

  unit_test::test_product_tensor_legendre< KokkosArray::SparseProductTensorLegendre< float , device > , float , float >( var , ngrid , 1 , true );
  unit_test::test_product_tensor_legendre< KokkosArray::SparseProductTensorLegendre< double, device > , double, double>( var , ngrid , 1 , true );

  //------------------------------------

  std::cout << std::endl << "\"Cuda Performance\"" << std::endl ;

  unit_test::performance_test_driver<Scalar,KokkosArray::Cuda>::run(
    test_flat, test_orig, test_block, check);

  KokkosArray::Cuda::finalize();

  return 0 ;
}

template int mainCuda<float>(bool, bool, bool, bool, int);
template int mainCuda<double>(bool, bool, bool, bool, int);
