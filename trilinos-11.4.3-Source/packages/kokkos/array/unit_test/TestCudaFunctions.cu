#include "hip/hip_runtime.h"
/*
//@HEADER
// ************************************************************************
// 
//   KokkosArray: Manycore Performance-Portable Multidimensional Arrays
//              Copyright (2012) Sandia Corporation
// 
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions? Contact  H. Carter Edwards (hcedwar@sandia.gov) 
// 
// ************************************************************************
//@HEADER
*/

#include <gtest/gtest.h>

#include <iostream>

#include <KokkosArray_Host.hpp>
#include <KokkosArray_Cuda.hpp>

#include <KokkosArray_View.hpp>
#include <KokkosArray_CrsArray.hpp>


//----------------------------------------------------------------------------

#include <TestViewImpl.hpp>
#include <TestAtomic.hpp>

#include <TestViewAPI.hpp>
#include <TestCrsArray.hpp>

#include <TestReduce.hpp>
#include <TestMultiReduce.hpp>

namespace Test {

__global__
void test_abort()
{
  KokkosArray::VerifyExecutionSpaceCanAccessDataSpace<
    KokkosArray::CudaSpace ,
    KokkosArray::HostSpace >::verify();
}


void test_device_cuda_view_impl()
{
  // test_abort<<<32,32>>>(); // Aborts the kernel with CUDA version 4.1 or greater

  test_view_impl< KokkosArray::Cuda >();
}

void test_device_cuda_view_api()
{
  TestViewAPI< double , KokkosArray::Cuda >();

#if 0
  KokkosArray::View<double, KokkosArray::Cuda > x("x");
  KokkosArray::View<double[1], KokkosArray::Cuda > y("y");
  // *x = 10 ;
  // x() = 10 ;
  // y[0] = 10 ;
  // y(0) = 10 ;
#endif
}

void test_device_cuda_crsarray() {
  TestCrsArray< KokkosArray::Cuda >();
}

void test_device_cuda_reduce() {
  TestReduce< long ,   KokkosArray::Cuda >( 10000000 );
  TestReduce< double , KokkosArray::Cuda >( 1000000 );
}

void test_device_cuda_reduce_dynamic() {
  TestReduceDynamic< long ,   KokkosArray::Cuda >( 10000000 );
  TestReduceDynamic< double , KokkosArray::Cuda >( 1000000 );
}

void test_device_cuda_reduce_dynamic_view() {
  TestReduceDynamicView< long ,   KokkosArray::Cuda >( 10000000 );
  TestReduceDynamicView< double , KokkosArray::Cuda >( 1000000 );
}

void test_device_cuda_multi_reduce() {
  TestReduceMulti< long , KokkosArray::Cuda >( 1000000 , 7 );
}

void test_device_cuda_atomic()
{
  const int loop_count = 1e3 ;

  ASSERT_TRUE( ( TestAtomic::Loop<int,KokkosArray::Cuda>(loop_count,1) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<int,KokkosArray::Cuda>(loop_count,2) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<int,KokkosArray::Cuda>(loop_count,3) ) );

  ASSERT_TRUE( ( TestAtomic::Loop<unsigned int,KokkosArray::Cuda>(loop_count,1) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<unsigned int,KokkosArray::Cuda>(loop_count,2) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<unsigned int,KokkosArray::Cuda>(loop_count,3) ) );

  ASSERT_TRUE( ( TestAtomic::Loop<long int,KokkosArray::Cuda>(loop_count,1) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<long int,KokkosArray::Cuda>(loop_count,2) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<long int,KokkosArray::Cuda>(loop_count,3) ) );

  ASSERT_TRUE( ( TestAtomic::Loop<unsigned long int,KokkosArray::Cuda>(loop_count,1) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<unsigned long int,KokkosArray::Cuda>(loop_count,2) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<unsigned long int,KokkosArray::Cuda>(loop_count,3) ) );

  ASSERT_TRUE( ( TestAtomic::Loop<long long int,KokkosArray::Cuda>(loop_count,1) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<long long int,KokkosArray::Cuda>(loop_count,2) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<long long int,KokkosArray::Cuda>(loop_count,3) ) );

  ASSERT_TRUE( ( TestAtomic::Loop<double,KokkosArray::Cuda>(loop_count,1) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<double,KokkosArray::Cuda>(loop_count,2) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<double,KokkosArray::Cuda>(loop_count,3) ) );

  ASSERT_TRUE( ( TestAtomic::Loop<float,KokkosArray::Cuda>(100,1) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<float,KokkosArray::Cuda>(100,2) ) );
  ASSERT_TRUE( ( TestAtomic::Loop<float,KokkosArray::Cuda>(100,3) ) );
}

}
